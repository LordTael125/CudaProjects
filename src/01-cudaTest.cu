#include <stdio.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>


// Macro for checking CUDA errors
#define CHECK_CUDA_ERROR(call)                                      \
    do {                                                            \
        hipError_t err = call;                                     \
        if (err != hipSuccess) {                                   \
            fprintf(stderr, "CUDA Error: %s (%s:%d)\n",             \
                    hipGetErrorString(err), __FILE__, __LINE__);   \
            exit(EXIT_FAILURE);                                     \
        }                                                           \
    } while (0)



struct bodyCar{
    float3 pos;
    float3 vel;
} typedef bodyCar;


    // A program to simulate total position change over time
__global__ void moveBody(bodyCar *body, int len, int dt){

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < len){
        body[i].pos.x += body[i].vel.x * dt;
        body[i].pos.y += body[i].vel.y * dt;
        body[i].pos.z += body[i].vel.z * dt;

    }
    

}

__global__ void initRNG(hiprandState *states, unsigned long seed){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed + i, i ,10, &states[i]);

}

__global__ void initCars(bodyCar *body, hiprandState *states, int len){

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < len){

        hiprandState localstate = states[i];

        body[i].pos.x = hiprand_uniform(&localstate) * 100.0;
        body[i].pos.y = hiprand_uniform(&localstate) * 100.0;
        body[i].pos.z = hiprand_uniform(&localstate) * 100.0;

        body[i].vel.x = hiprand_uniform(&localstate) * 10.0;
        body[i].vel.y = hiprand_uniform(&localstate) * 10.0;
        body[i].vel.z = hiprand_uniform(&localstate) * 10.0;

        states[i] = localstate;

        if (i < 10) {
            printf("Generated pos: %f, %f, %f\n", body[i].pos.x, body[i].pos.y, body[i].pos.z);
        }

    }
}



int main(){

    int NUM = 329;
    int my_time = 5;

    bodyCar Cars[NUM];

    bodyCar *d_body;
    hiprandState *d_states;


    
    CHECK_CUDA_ERROR(hipMalloc(&d_body, NUM * sizeof(bodyCar)));
    CHECK_CUDA_ERROR(hipMalloc(&d_states, NUM * sizeof(hiprandState)));

    int ThreadPerBlocks = 128;
    int BlocksPerGrid = (NUM + ThreadPerBlocks - 1) / ThreadPerBlocks;

    initRNG<<< BlocksPerGrid, ThreadPerBlocks>>>(d_states, time(NULL));
    
    initCars<<< BlocksPerGrid, ThreadPerBlocks>>>(d_body, d_states, NUM);
    
    moveBody<<< BlocksPerGrid, ThreadPerBlocks>>>(d_body, NUM, 45);
    
    
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipMemcpy(Cars, d_body, NUM * sizeof(bodyCar), hipMemcpyDeviceToHost));


    srand(time(NULL));
    for (int i = 0; i < 10; ++i) {
        int idx = rand() % 329;
        printf("pos after 45s - Car[%d].pos.x = %f\n", idx, Cars[idx].pos.x);
    }
    

    printf("pos: %f, %f, %f\n", Cars[0].pos.x, Cars[2].pos.y, Cars[8].pos.z);



    hipFree(d_body);
    hipFree(d_states);

    printf("Exec complete");
    return 0;
}